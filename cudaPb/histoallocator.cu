#include "histoallocator.h"
#include <cstdlib>
#include <stdio.h>


HistoAllocator::HistoAllocator(int width, int height, int arcno, int scale): 
	m_Width(width), m_Height(height), m_ArcNo(arcno), m_Scale(scale) {

	m_HistoCellSize = 256 * 2 * m_ArcNo * (m_Width + 2 * m_Scale) * sizeof(unsigned int);
	//find how much memory is available
	size_t total = 0;
	size_t free = 0;
	m_LastCudaError = hipMemGetInfo(&free, &total);

	if (m_LastCudaError != hipSuccess) 
		return;
	

	m_NoHistoChunks = free / 4 / m_HistoCellSize;
	if (m_Scale * 4 < m_NoHistoChunks)
		m_NoHistoChunks = m_Scale * 4;
	//printf("Allocating 2 chunks with %zu histo cells. Free %zu Total %zu\n", m_NoHistoChunks, free, total);
	//printf("Arcno %d Width %d Scale %d\n", m_ArcNo, m_Width, m_Scale);
	//printf("Cell size %zu %llu\n", m_HistoCellSize, sizeof(unsigned int));

	//preparing histograms
	m_LastCudaError = hipMalloc((void**)&m_dHistograms, 2 * sizeof(unsigned int*));

	if (m_LastCudaError != hipSuccess) {
		return;
	}

	m_hHistograms = (unsigned int**)malloc(2 * sizeof(unsigned int*));

	for (int i = 0; i < 2; ++i) {
		m_LastCudaError = hipMalloc((void**)&m_hHistograms[i], m_NoHistoChunks * m_HistoCellSize);
		//printf("Alloc %d\n", i);
		if (m_LastCudaError != hipSuccess) {
			printf("hipMalloc error 1: %d\n", i);
			return;
		}

		hipMemcpy(m_dHistograms, m_hHistograms, 2 * sizeof(unsigned int*), hipMemcpyHostToDevice);
		if (m_LastCudaError != hipSuccess) {
			printf("hipMemcpy error 1\n");
			return;
		}
	}

	m_LastCudaError = hipMemset(m_hHistograms[0], 0, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		printf("hipMemset error 1\n");
		return;
	}

	m_LastCudaError = hipMemset(m_hHistograms[1], 0, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		printf("hipMemset error 2\n");
		return;
	}

	m_TopChunk1 = int(m_NoHistoChunks);
	m_BottomChunk1 = 0;
	m_TopChunk2 = 2 * int(m_NoHistoChunks);
	m_BottomChunk2 = int(m_NoHistoChunks);
}


HistoAllocator::~HistoAllocator() {
	hipFree(m_hHistograms[0]);
	hipFree(m_hHistograms[1]);
	hipFree(m_dHistograms);
	free(m_hHistograms);
}

void HistoAllocator::setNewTopChunk() {	
	if (m_BottomChunk1 < m_BottomChunk2) {
		m_LastCudaError = hipMemset(m_hHistograms[0], 0, m_NoHistoChunks * m_HistoCellSize);
		if (m_LastCudaError != hipSuccess) {
			return;
		}
		m_BottomChunk1 = m_TopChunk2;
		m_TopChunk1 = m_TopChunk2 + m_TopChunk2 - m_BottomChunk2;
	}
	else {
		m_LastCudaError = hipMemset(m_hHistograms[1], 0, m_NoHistoChunks * m_HistoCellSize);
		if (m_LastCudaError != hipSuccess) {
			return;
		}
		m_BottomChunk2 = m_TopChunk1;
		m_TopChunk2 = m_TopChunk1 + m_TopChunk1 - m_BottomChunk1;
	}
}