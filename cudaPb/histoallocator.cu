#include "histoallocator.h"
#include <cstdlib>
#include <stdio.h>

HistoAllocator::HistoAllocator(int width, int height, int arcno, int scale)
	: m_Width(width), m_Height(height), m_ArcNo(arcno), m_Scale(scale),
	m_HistoCellSize(256 * 2 * m_ArcNo * (m_Width + 2 * m_Scale) * sizeof(unsigned int))
{
	//find how much memory is available
	int total = 0;
	int free = 0;
	m_LastCudaError = hipMemGetInfo((size_t*)&free, (size_t*)&total);

	if (m_LastCudaError != hipSuccess)
		return;

	int m_NoHistoChunks = free / 4 / m_HistoCellSize;
	printf("Allocating 2 chunks with %d histo cells\n", m_NoHistoChunks);

	m_LastCudaError = hipMalloc((void**)&m_dChunk1, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		return;
	}

	m_LastCudaError = hipMemset(m_dChunk1, 0, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		return;
	}

	m_LastCudaError = hipMalloc((void**)&m_dChunk2, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		return;
	}

	m_LastCudaError = hipMemset(m_dChunk2, 0, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		return;
	}

	//TODO: hipMemset

	m_TopChunk1 = m_NoHistoChunks;
	m_BottomChunk1 = 0;
	m_TopChunk2 = 2 * m_NoHistoChunks;
	m_BottomChunk2 = m_NoHistoChunks;
}


HistoAllocator::~HistoAllocator() {
	hipFree(m_dChunk1);
	hipFree(m_dChunk2);
}

void HistoAllocator::setNewTopChunk() {	
	printf("SetNewTopChunk\n");
	m_LastCudaError = hipFree(m_dChunk1);
	if (m_LastCudaError != hipSuccess) {
		printf("BlaBla1\n");
		//return;
	}

	unsigned int* temp;

	m_LastCudaError = hipMalloc((void**)&temp, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		printf("BlaBla2\n");
		return;
	}

	m_LastCudaError = hipMemset(temp, 0, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		printf("BlaBla3\n");
		return;
	}

	//TODO: is this really working??
	m_dChunk1 = m_dChunk2;
	m_dChunk2 = temp;

	m_BottomChunk1 = m_BottomChunk2;
	m_TopChunk1 = m_TopChunk2;
	m_BottomChunk2 = m_TopChunk1;
	m_TopChunk2 = m_BottomChunk2 + m_TopChunk1 - m_BottomChunk1;
}