#include "histoallocator.h"
#include <cstdlib>
#include <stdio.h>

HistoAllocator::HistoAllocator(int width, int height, int arcno, int scale)
	: m_Width(width), m_Height(height), m_ArcNo(arcno), m_Scale(scale),
	m_HistoCellSize(256 * 2 * m_ArcNo * (m_Width + 2 * m_Scale) * sizeof(unsigned int))
{
	//find how much memory is available
	size_t total = 0;
	size_t free = 0;
	m_LastCudaError = hipMemGetInfo(&free, &total);

	if (m_LastCudaError != hipSuccess)
		return;

	size_t m_NoHistoChunks = free / 4 / m_HistoCellSize;
	printf("Allocating 2 chunks with %zu histo cells. Free %zu Total %zu\n", m_NoHistoChunks, free, total);

	m_LastCudaError = hipMalloc((void**)&m_dChunk1, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		return;
	}

	m_LastCudaError = hipMemset(m_dChunk1, 0, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		return;
	}

	m_LastCudaError = hipMalloc((void**)&m_dChunk2, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		return;
	}

	m_LastCudaError = hipMemset(m_dChunk2, 0, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		return;
	}

	//TODO: hipMemset

	m_TopChunk1 = int(m_NoHistoChunks);
	m_BottomChunk1 = 0;
	m_TopChunk2 = 2 * int(m_NoHistoChunks);
	m_BottomChunk2 = int(m_NoHistoChunks);
}


HistoAllocator::~HistoAllocator() {
	hipFree(m_dChunk1);
	hipFree(m_dChunk2);
}

void HistoAllocator::setNewTopChunk() {	
	printf("SetNewTopChunk\n");
	/*m_LastCudaError = hipFree(m_dChunk1);
	if (m_LastCudaError != hipSuccess) {
		printf("BlaBla1\n");
		//return;
	}*/

	/*unsigned int* temp;

	m_LastCudaError = hipMalloc((void**)&temp, m_NoHistoChunks * m_HistoCellSize);
	if (m_LastCudaError != hipSuccess) {
		printf("BlaBla2\n");
		return;
	}*/

	if (m_BottomChunk1 < m_BottomChunk2) {
		m_LastCudaError = hipMemset(m_dChunk1, 0, m_NoHistoChunks * m_HistoCellSize);
		if (m_LastCudaError != hipSuccess) {
			printf("BlaBla3\n");
			return;
		}
		m_BottomChunk1 = m_TopChunk2;
		m_TopChunk1 = m_TopChunk2 + m_TopChunk2 - m_BottomChunk2;
	}
	else {
		m_LastCudaError = hipMemset(m_dChunk2, 0, m_NoHistoChunks * m_HistoCellSize);
		if (m_LastCudaError != hipSuccess) {
			printf("BlaBla3\n");
			return;
		}
		m_BottomChunk2 = m_TopChunk1;
		m_TopChunk2 = m_TopChunk1 + m_TopChunk1 - m_BottomChunk1;
	}
}