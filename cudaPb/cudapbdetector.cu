#include "hip/hip_runtime.h"
#include "cudapbdetector.h"
#include "cvector.h"
#include <cstdlib>
#include <thread>


__global__ void calculateGradients(int row_start, int row_count, double* dGradientImages, unsigned int** dHistograms, int image_width, int image_height, int scale, int arcno) {
	int row = row_start + blockIdx.x;
	int index = threadIdx.x;
	int stride = blockDim.x;
	
	if (row < 2 * scale || row >= image_height + 2 * scale || row >= row_start + row_count)
		return;

	for (int j = scale + index; j < image_width + scale; j += stride) {
		unsigned int* vHist = dHistograms[row - scale] +  j * 2 * arcno * 256;

		for (int i = 0; i < arcno; ++i) {
			unsigned int* histo1 = vHist + i * 256;
			unsigned int* histo2 = vHist + (i + arcno) * 256;
			//printf("Chi square for:\n");
			double val = 0.0;
			for (int k = 0; k < 256; ++k) {
				if (histo1[k] != 0 || histo2[k] != 0) {
					double diff = double(int(histo1[k] - histo2[k]));
					double sum = double(histo1[k] + histo2[k]);
					val = val + diff * diff / sum;
					//printf("[%d - %d = %f]", histo1[k], histo2[k], val);
				}

			}
			double grad = val;
			//printf("Grad[%d, %d, %d]=%f\n", row, j, i, grad);
			*(dGradientImages + i * image_width * image_height + (row - 2 * scale) * image_width + (j - scale)) = grad;
		}
	}
}

__global__ void calcHisto(int row_start, int row_count, unsigned char* dSourceImage, struct CVector* dHalfDiscInfluencePoints, int totalHalfInfluencePoints, unsigned int** dHistograms, int image_width, int image_height, int scale, int arcno)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	int i = row_start + blockIdx.x;

	if (blockIdx.x > row_count)
		return;

	for (int j = index; j < image_width + 2 * scale; j += stride) {
		//qDebug() << "BlaBla1 " << j;
		unsigned char val = dSourceImage[i * (image_width + 2 * scale) + j];
		//printf("Index %d Val %d \n", j, int(val));
		//with the point (i,j) with value val, update all histograms which contain this data point
		addToHistoArray(dHalfDiscInfluencePoints, totalHalfInfluencePoints, dHistograms, image_width, image_height, scale, arcno, val, i, j);
	}
}

__device__ void addToHistoArray(struct CVector* dHalfDiscInfluencePoints, int totalHalfInfluencePoints, unsigned int** dHistograms, int image_width, int image_height, int scale, int arcno, int val, int i, int j)
{
	for (int k = 0; k < totalHalfInfluencePoints; ++k) {
		struct CVector n = dHalfDiscInfluencePoints[k];
		if ((n.m_Data[0] + i) < 0 || (n.m_Data[0] + i) >= image_height + 2 * scale)
			continue;
		if ((n.m_Data[1] + j) < 0 || (n.m_Data[1] + j) >= image_width + 2 * scale)
			continue;

		//qDebug() << "Compute at " << i << " with " << (n[0] + i) << " size " << vMaps[n[0] + i].size();
		/*if (int(vMaps[n[0] + i].size()) != m_SingleChannelImage.cols + 2 * m_Scale) {
			qDebug() << "exiting ..";
			exit(1);
		}*/

		unsigned int* vHist = dHistograms[n.m_Data[0] + i] + (n.m_Data[1] + j) * 2 * arcno * 256;
		for (unsigned int l = 2; l < n.m_Size; ++l) {
			if (n.m_Data[l] > 2 * arcno)
				continue;
			//qDebug() << "Insert into histo " << n[k] << " val " << val << " vHist size " << vHist.size();
			//TODO: use atomic operation
			atomicInc(vHist + n.m_Data[l] * 256 + val, 4 * scale * scale);
		}
	}
}

CudaPbDetector::CudaPbDetector(unsigned char* image_data, int image_width, int image_height, int scale) :
	m_Width(image_width), m_Height(image_height), m_Scale(scale)
{
	if (!createGradientImages()) {
		printf("Error in constructor createGradientImages\n");
		return;
	}

	if (!copyImageToGPU(image_data)) {
		printf("Error in constructor copyImageToGPU\n");
		return;
	}

	if (!create2DHistoArray()) {
		printf("Error in constructor create2DHistoArray\n");
		return;
	}

	if (!initializeInfluencePoints()) {
		printf("Error in constructor initializeInfluencePoints\n");
		return;
	}

	hipStreamCreate(&m_Stream1_2);
	hipStreamCreate(&m_Stream1_1);
	hipStreamCreateWithFlags(&m_Stream2_1, hipStreamNonBlocking);
	hipStreamCreateWithFlags(&m_Stream2_2, hipStreamNonBlocking);

	m_FullyInitialized = true;
}

bool CudaPbDetector::createGradientImages()
{
	//allocate the device memory for the gradient images
	m_LastCudaError = hipMalloc(&m_dGradientImages, m_ArcNo * m_Width * m_Height * sizeof(double));

	if (m_LastCudaError != hipSuccess) {
		return false;
	}

	//set all pixels in the gradient images to 0
	m_LastCudaError = hipMemset(m_dGradientImages, 0, m_ArcNo * m_Width * m_Height * sizeof(double));

	if (m_LastCudaError != hipSuccess)
		return false;	

	m_hGradientImages = (double*)malloc(m_ArcNo * m_Width * m_Height * sizeof(double));

	return true;
}

bool CudaPbDetector::initializeHistoRange(int start, int stop)
{
	for (int i = start; i < stop; ++i) {
		//printf("Alloc %d\n", i);
		m_LastCudaError = hipMalloc((void**)&m_hHistograms[i],   256 * 2 * m_ArcNo * (m_Width + 2 * m_Scale) * sizeof(unsigned int));
		//printf("Alloc %d\n", i);
		if (m_LastCudaError != hipSuccess) {
			printf("hipMalloc error 1: %d - %s - %d\n", i, hipGetErrorString(m_LastCudaError), m_BottomAllocated);
			return false;
		}
	}

	hipMemcpy(m_dHistograms + start, m_hHistograms + start, (stop - start) * sizeof(unsigned int*), hipMemcpyHostToDevice);
	if (m_LastCudaError != hipSuccess) {
		printf("hipMemcpy error 1\n");
		return false;
	}
	
	return true;
}

bool CudaPbDetector::deleteFromHistoMaps(int index) {
	if (index >= m_Scale + 1) {
		m_LastCudaError = hipFree(m_hHistograms[index - m_Scale - 1]);
		//printf("Free %d \n", index - m_Scale - 1);
		if (m_LastCudaError != hipSuccess) {
			printf("hipFree error 1: %d - %s\n", index, hipGetErrorString(m_LastCudaError));
			return false;
		}
		//printf("Delete %d\n", index - m_Scale - 1);
		m_hHistograms[index - m_Scale - 1] = nullptr;
	}

	return true;
}

bool CudaPbDetector::create2DHistoArray()
{
	//preparing histograms
	m_LastCudaError = hipMalloc((void**)&m_dHistograms, (m_Height + 2 * m_Scale) * sizeof(unsigned int*));

	if (m_LastCudaError != hipSuccess)
		return false;

	m_hHistograms = (unsigned int**)malloc((m_Height + 2 * m_Scale) * sizeof(unsigned int*));

	return true;
}

//TODO: to check this
CudaPbDetector::~CudaPbDetector()
{
	hipFree(m_dSourceImage);
	hipFree(m_dGradientImages);
	
	for (int i = 0; i < m_Height + 2 * m_Scale; ++i) {
		if (m_hHistograms[i])
			hipFree(m_hHistograms[i]);
	}

	free(m_hHistograms);
	hipFree(m_dHistograms);

	for (int i = 0; i < m_TotalHalfInfluencePoints; ++i) {
		hipFree(m_hHalfDiscInfluencePoints[i].m_Data); 
	}

	free(m_hHalfDiscInfluencePoints);
	hipFree(m_dHalfDiscInfluencePoints);

	hipStreamDestroy(m_Stream1_2);
	hipStreamDestroy(m_Stream1_1);
	hipStreamDestroy(m_Stream2_1);
	hipStreamDestroy(m_Stream2_1);
}

bool CudaPbDetector::copyImageToGPU(unsigned char* image_data)
{
	//copy image to the device memory and pad with zeros
	//TODO: for start pad with zeros, but later as in the CPU method
	//another solution is to pad the image before it is given to this class
	m_LastCudaError = hipMalloc((void**)&m_dSourceImage, (m_Width + 2 * m_Scale) * (m_Height + 2 * m_Scale));

	if (m_LastCudaError != hipSuccess) {
		printf("Error copyImageToGPU hipMalloc %s, %d %d %d\n", hipGetErrorString(m_LastCudaError), m_Width, m_Height, m_Scale);
		//return false;
	}

	//set all pixels in the image to zero 
	m_LastCudaError = hipMemset(m_dSourceImage, 0, (m_Width + 2 * m_Scale) * (m_Height + 2 * m_Scale));

	if (m_LastCudaError != hipSuccess) {
		printf("Error copyImageToGPU hipMemset\n");
		return false;
	}

	//copy from the host image with padding
	int count = 0;
	while (count < m_Height && m_LastCudaError == hipSuccess)
	{
		m_LastCudaError = hipMemcpy(m_dSourceImage + m_Scale * (m_Width + 2 * m_Scale) + count * (m_Width + 2 * m_Scale) + m_Scale, image_data + count * m_Width, m_Width, hipMemcpyHostToDevice);
		if (m_LastCudaError != hipSuccess) {
			printf("Error copyImageToGPU hipMemcpy %d\n", count);
			return false;
		}
		count++;
	}
	
	return true;
}

/**
 * Copies m_Masks->getHalfDiscInfluencePoints()
 * to the GPU
 */
bool CudaPbDetector::initializeInfluencePoints() {
	m_Masks = new DiscInverseMasks(m_Scale);
	std::vector<std::vector<int>> neighb = m_Masks->getHalfDiscInfluencePoints();

	m_TotalHalfInfluencePoints = int(neighb.size());

	m_hHalfDiscInfluencePoints = (CVector*)malloc(m_TotalHalfInfluencePoints * sizeof(CVector));
	for (int i = 0; i < m_TotalHalfInfluencePoints; ++i) {
		m_hHalfDiscInfluencePoints[i].m_Size = int(neighb[i].size());
		m_LastCudaError = hipMalloc(&m_hHalfDiscInfluencePoints[i].m_Data, neighb[i].size() * sizeof(int));
		if (m_LastCudaError != hipSuccess)
			return false;
		int* values = (int*)malloc(neighb[i].size() * sizeof(int));
		
		for (int j = 0; j < neighb[i].size(); ++j)
			values[j] = neighb[i][j];
		m_LastCudaError = hipMemcpy(m_hHalfDiscInfluencePoints[i].m_Data, values, neighb[i].size() * sizeof(int), hipMemcpyHostToDevice);
		if (m_LastCudaError != hipSuccess)
			return false;
		free(values);
	}

//TODO: release memory in case of failure

	//preparing histograms
	m_LastCudaError = hipMalloc(&m_dHalfDiscInfluencePoints, m_TotalHalfInfluencePoints * sizeof(CVector));
	if (m_LastCudaError != hipSuccess)
		return false;

	hipMemcpy(m_dHalfDiscInfluencePoints, m_hHalfDiscInfluencePoints, m_TotalHalfInfluencePoints * sizeof(CVector), hipMemcpyHostToDevice);
	if (m_LastCudaError != hipSuccess)
		return false;

	return true;	
}

bool CudaPbDetector::executeStreaming() {
	if (!initializeHistoRange(0, m_Scale + 1))
		return false;

	m_TopAllocated = m_Scale + 1;
	m_BottomAllocated = 0;

	std::thread producer(&CudaPbDetector::producerThread, this);
	std::thread consumer(&CudaPbDetector::consumerThread, this);

	producer.join();
	consumer.join();

	m_LastCudaError = hipMemcpy(m_hGradientImages, m_dGradientImages, m_ArcNo * m_Width * m_Height * sizeof(double), hipMemcpyDeviceToHost);
	return m_LastCudaError == hipSuccess;	
}

void CudaPbDetector::producerThread() {

	m_TopAllocated = 0;
	m_BottomAllocated = 0;

	//new loop
	while (m_TopAllocated < m_Height + m_Scale - 1) {
		//if (m_TopAllocated >= 100)
		//	exit(1);

		int row_start = m_TopAllocated;
		int row_count = std::min(m_Step, m_Height + 2 * m_Scale - m_TopAllocated);
		//printf("Row_start: %d Row_count %d \n", row_start, row_count);

		int new_top_allocated = m_TopAllocated;
//		bool allocate_failed = false;

		//m_HistoMutex.lock();
		for (int k = row_start; k < row_count + row_start; ++k) {
			if (k + m_Scale + 1 < m_Height + 2 * m_Scale) {
				if (initializeHistoRange(k + m_Scale + 1, k + m_Scale + 2)) {
					new_top_allocated = k + 1;
				}
				else {
					//allocate_failed = true;
					row_count = new_top_allocated - m_TopAllocated;
					break;
				}
			}
		}
		//m_HistoMutex.unlock();

		if (row_count) {
			if (row_count != m_Step) {
				calcHisto << <row_count, m_NoThreads, 0, m_Stream1_2 >> > (row_start, row_count, m_dSourceImage, m_dHalfDiscInfluencePoints, m_TotalHalfInfluencePoints, m_dHistograms, m_Width, m_Height, m_Scale, m_ArcNo);
				//synchronize in stream
				hipStreamSynchronize(m_Stream1_2);
			}
			else {
				calcHisto << <row_count / 2, m_NoThreads, 0, m_Stream1_2 >> > (row_start, row_count / 2, m_dSourceImage, m_dHalfDiscInfluencePoints, m_TotalHalfInfluencePoints, m_dHistograms, m_Width, m_Height, m_Scale, m_ArcNo);
				//synchronize in stream
				calcHisto << <row_count / 2 , m_NoThreads, 0, m_Stream1_1 >> > (row_start + row_count / 2, row_count / 2, m_dSourceImage, m_dHalfDiscInfluencePoints, m_TotalHalfInfluencePoints, m_dHistograms, m_Width, m_Height, m_Scale, m_ArcNo);
				//synchronize in stream
				hipStreamSynchronize(m_Stream1_2);
				hipStreamSynchronize(m_Stream1_1);

			}
 			m_TopAllocated = new_top_allocated;
			//printf("m_TopAllocated = %d\n", m_TopAllocated);
		}
	}
}

void CudaPbDetector::consumerThread() {
	while (m_BottomAllocated + m_Scale + 1 < m_Height + 2 * m_Scale) {
		//to check
		if (m_BottomAllocated >= m_TopAllocated)
			continue;


		int row_start = m_BottomAllocated;
		int row_count = std::min(m_Step, m_TopAllocated - m_BottomAllocated);

		calculateGradients <<<row_count / 2, m_NoThreads, 0, m_Stream2_1>>> (row_start, row_count / 2, m_dGradientImages, m_dHistograms, m_Width, m_Height, m_Scale, m_ArcNo);
		calculateGradients << <row_count / 2, m_NoThreads, 0, m_Stream2_2 >> > (row_start + row_count / 2, row_count / 2, m_dGradientImages, m_dHistograms, m_Width, m_Height, m_Scale, m_ArcNo);
		//synchronize in stream
		hipStreamSynchronize(m_Stream2_1);
		hipStreamSynchronize(m_Stream2_2);

		//m_HistoMutex.lock();
		for (int k = row_start; k < row_count + row_start; ++k) {
			if (!deleteFromHistoMaps(k)) {
				printf("Error releasing CUDA memory\n");
				exit(1);
			}
		}
		//m_HistoMutex.unlock();

		m_BottomAllocated += row_count;
		//printf("m_BottomAllocated = %d \n", m_BottomAllocated);
	}
}