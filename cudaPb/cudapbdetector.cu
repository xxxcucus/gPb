#include "hip/hip_runtime.h"
#include "cudapbdetector.h"
#include "cvector.h"
#include <cstdlib>


__global__ void calculateGradients(int row_start, int row_count, double* dGradientImages, \
	unsigned int** dHistograms, int bottomChunk1, int bottomChunk2, int topChunk1, int topChunk2,\
	int image_width, int image_height, int scale, int arcno) {
	int row = row_start + blockIdx.x;
	int index = threadIdx.x;
	int stride = blockDim.x;
	
	if (row < 2 * scale || row >= image_height + 2 * scale || row >= row_start + row_count)
		return;

	for (int j = scale + index; j < image_width + scale; j += stride) {
		//unsigned int* vHist = dHistograms[row - scale] +  j * 2 * arcno * 256;
		unsigned int* vHist = getHistoPointer(row - scale, j, dHistograms, bottomChunk1, bottomChunk2, topChunk1, topChunk2, image_width, scale, arcno);
		
		//todo: error handling
		if (vHist == nullptr)
			return;

		for (int i = 0; i < arcno; ++i) {
			unsigned int* histo1 = vHist + i * 256;
			unsigned int* histo2 = vHist + (i + arcno) * 256;
			//printf("Chi square for:\n");
			double val = 0.0;
			for (int k = 0; k < 256; ++k) {
				if (histo1[k] != 0 || histo2[k] != 0) {
					double diff = double(int(histo1[k] - histo2[k]));
					double sum = double(histo1[k] + histo2[k]);
					val = val + diff * diff / sum;
					//printf("[%d - %d = %f]", histo1[k], histo2[k], val);
				}

			}
			double grad = val;
			//printf("Grad[%d, %d, %d]=%f\n", row, j, i, grad);
			*(dGradientImages + i * image_width * image_height + (row - 2 * scale) * image_width + (j - scale)) = grad;
		}
	}
}

__global__ void calcHisto(int row_start, int row_count, unsigned char* dSourceImage,\
	int* dHalfDiscInfluencePoints, int totalHalfInfluencePoints, \
	unsigned int** dHistograms, int bottomChunk1, int bottomChunk2, int topChunk1, int topChunk2,\
	int image_width, int image_height, int scale, int arcno)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	int i = row_start + blockIdx.x;

	if (blockIdx.x > row_count)
		return;

	for (int j = index; j < image_width + 2 * scale; j += stride) {
		//qDebug() << "BlaBla1 " << j;
		unsigned char val = dSourceImage[i * (image_width + 2 * scale) + j];
		//printf("Row %d Index %d Val %d \n", i, j, int(val));
		//with the point (i,j) with value val, update all histograms which contain this data point
		addToHistoArray(dHalfDiscInfluencePoints, totalHalfInfluencePoints, dHistograms, bottomChunk1, bottomChunk2, topChunk1, topChunk2, image_width, image_height, scale, arcno, val, i, j);
	}
}

__device__ void addToHistoArray(int* dHalfDiscInfluencePoints, int totalHalfInfluencePoints,\
	unsigned int** dHistograms, int bottomChunk1, int bottomChunk2, int topChunk1, int topChunk2, \
	int image_width, int image_height, int scale, int arcno, int val, int i, int j)
{
	int data_size = 1;
	for (int k = 0; k < totalHalfInfluencePoints; k += data_size + 1) {
		data_size = dHalfDiscInfluencePoints[k];
		int* data = dHalfDiscInfluencePoints + k + 1; 
		int row = data[0] + i;
		int col = data[1] + j;
		if (row < 0 || row >= image_height + 2 * scale)
			continue;
		if (col < 0 || col >= image_width + 2 * scale)
			continue;

		//printf("Compute at  %d %d\n", n.m_Data[0] + i, n.m_Data[1] + j);
		/*if (int(vMaps[n[0] + i].size()) != m_SingleChannelImage.cols + 2 * m_Scale) {
			qDebug() << "exiting ..";
			exit(1);
		}*/

		//unsigned int* vHist = dHistograms[n.m_Data[0] + i] + (n.m_Data[1] + j) * 2 * arcno * 256;
		unsigned int* vHist = getHistoPointer(row, col, dHistograms, bottomChunk1, bottomChunk2, topChunk1, topChunk2, image_width, scale, arcno);
		//todo: error handling
		if (vHist == nullptr)
			continue;
		
		for (unsigned int l = 2; l < dHalfDiscInfluencePoints[k]; ++l) {
			int idx = data[l];
			if (idx >= 2 * arcno || val < 0 || val >= 256)
				continue;
			//qDebug() << "Insert into histo " << n[k] << " val " << val << " vHist size " << vHist.size();
			//TODO: use atomic operation
			atomicInc(vHist + idx * 256 + val, 4 * scale * scale);
		}
	}
}

__device__ unsigned int* getHistoPointer(int row, int col, \
	unsigned int** dHistograms, int bottomChunk1, int bottomChunk2, int topChunk1, int topChunk2,\
	int width, int scale, int arcno) {
	
	int bottomChunk = bottomChunk1;
	if (bottomChunk2 < bottomChunk1)
		bottomChunk = bottomChunk2;
	int topChunk = topChunk1;
	if (topChunk2 > topChunk1)
		topChunk = topChunk2;

	//printf("Bottom1 %d - Top1 %d\n Bottom2 %d - Top2 %d \n", bottomChunk1, topChunk1, bottomChunk2, topChunk2);
	//printf("Bottom %d - Top %d \n", bottomChunk, topChunk);
	
	if (row < bottomChunk || row >= topChunk)
		return nullptr;

	if (col < 0 || col >= width + 2 * scale)
		return nullptr;

	if (topChunk1 - bottomChunk1 != topChunk2 - bottomChunk2) {
		printf("Error difference 1\n");
		return nullptr;
	}

	if (topChunk - bottomChunk != 2 * (topChunk1 - bottomChunk1)) {
		printf("Error difference 2\n");
		return nullptr;
	}

	unsigned int* rowp;
	int middleChunk = (bottomChunk + topChunk) / 2;
	//printf("Row %d Middlechunk %d\n", row, middleChunk);
	if (row < middleChunk) {
		if ((row - bottomChunk < 0) || (row - bottomChunk >= topChunk1 - bottomChunk1)) {
			printf("Error bottomChunk!!!!\n");
			return nullptr;
		}
		long int d = 256 * 2 * long int(arcno) * long int(width + 2 * scale) * long int(row - bottomChunk);
		//printf("Offset %ld  Arcno %d Width %d Scale %d Factor %d\n", d, arcno, width, scale, row - bottomChunk);
		if (bottomChunk1 < bottomChunk2)
			rowp = dHistograms[0] + d;
		else
			rowp = dHistograms[1] + d;
	}
	else {
		if ((row - middleChunk < 0) || (row - middleChunk >= topChunk1 - bottomChunk1 - 1)) {
			printf("Error middleChunk!!!!\n");
			return nullptr;
		}
		long int d = 256 * 2 * long int(arcno) * long int(width + 2 * scale) * long int(row - middleChunk);

		if (bottomChunk1 < bottomChunk2)
			rowp = dHistograms[1] + d;
		else
			rowp = dHistograms[0] + d;
	}

	return rowp + 256 * 2 * arcno * col;
}


CudaPbDetector::CudaPbDetector(unsigned char* image_data, int image_width, int image_height, int scale) :
	m_Width(image_width), m_Height(image_height), m_Scale(scale)
{
	if (!createGradientImages()) {
		printf("Error in constructor createGradientImages\n");
		return;
	}

	if (!copyImageToGPU(image_data)) {
		printf("Error in constructor copyImageToGPU\n");
		return;
	}

	/*if (!create2DHistoArray()) {
		printf("Error in constructor create2DHistoArray\n");
		return;
	}*/

	if (!initializeInfluencePoints()) {
		printf("Error in constructor initializeInfluencePoints\n");
		return;
	}

	printf("Constructing histo allocator\n");
	m_HistoAllocator = new HistoAllocator(m_Width, m_Height, m_ArcNo, m_Scale, m_Step);
	if (m_HistoAllocator->wasError())
		return;

	readDeviceProperties();

	m_FullyInitialized = true;
}

bool CudaPbDetector::createGradientImages()
{
	//allocate the device memory for the gradient images
	m_LastCudaError = hipMalloc(&m_dGradientImages, m_ArcNo * m_Width * m_Height * sizeof(double));

	if (m_LastCudaError != hipSuccess) {
		return false;
	}

	//set all pixels in the gradient images to 0
	m_LastCudaError = hipMemset(m_dGradientImages, 0, m_ArcNo * m_Width * m_Height * sizeof(double));

	if (m_LastCudaError != hipSuccess)
		return false;	

	m_hGradientImages = (double*)malloc(m_ArcNo * m_Width * m_Height * sizeof(double));

	return true;
}

bool CudaPbDetector::initializeHistoRange(int start, int stop)
{
	//printf("InitializeHistoRange %d-%d\n", start, stop);
	int topChunk = std::max(m_HistoAllocator->m_TopChunk1, m_HistoAllocator->m_TopChunk2);
	int bottomChunk = std::min(m_HistoAllocator->m_BottomChunk1, m_HistoAllocator->m_BottomChunk2);
	
	if (stop < topChunk && start >= bottomChunk) {
		return true;
	}

	int middleChunk = (topChunk + bottomChunk) / 2;

	if (start < middleChunk) {
		printf("Error start below middleChunk\n");
		return false;
	}

	//TODO: when stop > m_HistoAllocator->m_TopChunk2 we must be sure that we do not need the bottom chunk anymore!!! - be carefull streaming

	m_HistoAllocator->setNewTopChunk();

	if (m_HistoAllocator->wasError()) {
		printf("Error when allocating new chunk - %s\n", hipGetErrorString(m_HistoAllocator->getError()));
		return false;
	}
	
	return true;
}

bool CudaPbDetector::updateHistoBuffer(int step, int index) {

	if (index + m_Scale + step + 1 < m_Height + 2 * m_Scale) {
		if (!initializeHistoRange(index + m_Scale + step + 1, index + m_Scale + step + 2))
			return false;
	}

	return true;
}

bool CudaPbDetector::create2DHistoArray()
{
	//preparing histograms
	m_LastCudaError = hipMalloc((void**)&m_dHistograms, (m_Height + 2 * m_Scale) * sizeof(unsigned int*));

	if (m_LastCudaError != hipSuccess)
		return false;

	m_hHistograms = (unsigned int**)malloc((m_Height + 2 * m_Scale) * sizeof(unsigned int*));

	return true;
}

//TODO: to check this
CudaPbDetector::~CudaPbDetector()
{
	hipFree(m_dSourceImage);
	hipFree(m_dGradientImages);

	delete m_HistoAllocator;

	free(m_hHalfDiscInfluencePoints);
	hipFree(m_dHalfDiscInfluencePoints);
}

bool CudaPbDetector::copyImageToGPU(unsigned char* image_data)
{
	//copy image to the device memory and pad with zeros
	//TODO: for start pad with zeros, but later as in the CPU method
	//another solution is to pad the image before it is given to this class
	m_LastCudaError = hipMalloc((void**)&m_dSourceImage, (m_Width + 2 * m_Scale) * (m_Height + 2 * m_Scale));

	if (m_LastCudaError != hipSuccess) {
		printf("Error copyImageToGPU hipMalloc %s, %d %d %d\n", hipGetErrorString(m_LastCudaError), m_Width, m_Height, m_Scale);
		//return false;
	}

	//set all pixels in the image to zero 
	m_LastCudaError = hipMemset(m_dSourceImage, 0, (m_Width + 2 * m_Scale) * (m_Height + 2 * m_Scale));

	if (m_LastCudaError != hipSuccess) {
		printf("Error copyImageToGPU hipMemset\n");
		return false;
	}

	//copy from the host image with padding
	int count = 0;
	while (count < m_Height && m_LastCudaError == hipSuccess)
	{
		m_LastCudaError = hipMemcpy(m_dSourceImage + m_Scale * (m_Width + 2 * m_Scale) + count * (m_Width + 2 * m_Scale) + m_Scale, image_data + count * m_Width, m_Width, hipMemcpyHostToDevice);
		if (m_LastCudaError != hipSuccess) {
			printf("Error copyImageToGPU hipMemcpy %d\n", count);
			return false;
		}
		count++;
	}
	
	return true;
}

/**
 * Copies m_Masks->getHalfDiscInfluencePoints()
 * to the GPU
 */
/*bool CudaPbDetector::initializeInfluencePoints() {
	m_Masks = new DiscInverseMasks(m_Scale);
	std::vector<std::vector<int>> neighb = m_Masks->getHalfDiscInfluencePoints();

	m_TotalHalfInfluencePoints = int(neighb.size());

	m_hHalfDiscInfluencePoints = (CVector*)malloc(m_TotalHalfInfluencePoints * sizeof(CVector));
	for (int i = 0; i < m_TotalHalfInfluencePoints; ++i) {
		m_hHalfDiscInfluencePoints[i].m_Size = int(neighb[i].size());
		m_LastCudaError = hipMalloc(&m_hHalfDiscInfluencePoints[i].m_Data, neighb[i].size() * sizeof(int));
		if (m_LastCudaError != hipSuccess)
			return false;
		int* values = (int*)malloc(neighb[i].size() * sizeof(int));
		
		for (int j = 0; j < neighb[i].size(); ++j)
			values[j] = neighb[i][j];
		m_LastCudaError = hipMemcpy(m_hHalfDiscInfluencePoints[i].m_Data, values, neighb[i].size() * sizeof(int), hipMemcpyHostToDevice);
		if (m_LastCudaError != hipSuccess)
			return false;
		free(values);
	}

//TODO: release memory in case of failure

	//preparing histograms
	m_LastCudaError = hipMalloc(&m_dHalfDiscInfluencePoints, m_TotalHalfInfluencePoints * sizeof(CVector));
	if (m_LastCudaError != hipSuccess)
		return false;

	hipMemcpy(m_dHalfDiscInfluencePoints, m_hHalfDiscInfluencePoints, m_TotalHalfInfluencePoints * sizeof(CVector), hipMemcpyHostToDevice);
	if (m_LastCudaError != hipSuccess)
		return false;

	return true;	
}*/

bool CudaPbDetector::initializeInfluencePoints() {
	m_Masks = new DiscInverseMasks(m_Scale);
	std::vector<std::vector<int>> neighb = m_Masks->getHalfDiscInfluencePoints();

	//save the vectors inside a single linear vector
	std::vector<int> linear_neighb;
	for (auto nb : neighb) {
		linear_neighb.push_back(int(nb.size()));
		for (auto ne : nb)
			linear_neighb.push_back(ne);
	}

	m_TotalHalfInfluencePoints = int(linear_neighb.size());
	//printf("Total influence points %d\n", m_TotalHalfInfluencePoints);

	m_hHalfDiscInfluencePoints = (int*)malloc(m_TotalHalfInfluencePoints * sizeof(int));
	for (unsigned int i = 0; i < linear_neighb.size(); ++i) {
		//printf("[%d] = %d\n", i, linear_neighb[i]);
		m_hHalfDiscInfluencePoints[i] = linear_neighb[i];
	}

	//preparing histograms
	m_LastCudaError = hipMalloc(&m_dHalfDiscInfluencePoints, m_TotalHalfInfluencePoints * sizeof(int));
	if (m_LastCudaError != hipSuccess)
		return false;

	hipMemcpy(m_dHalfDiscInfluencePoints, m_hHalfDiscInfluencePoints, m_TotalHalfInfluencePoints * sizeof(int), hipMemcpyHostToDevice);
	if (m_LastCudaError != hipSuccess)
		return false;

	return true;
}

bool CudaPbDetector::executeChunk() {

	if (!initializeHistoRange(0, m_Scale + m_Step + 1))
		return false;
	int noSteps = (m_Height + 2 * m_Scale + m_Step - 1) / m_Step;

	for (int i = 0; i < noSteps; ++i) {
		int row_start = m_Step * i;
		int row_count = std::min(m_Step, m_Height + 2 * m_Scale - row_start);
		//printf("Row_start: %d Row_count %d Scale %d BottomChunk1 %d TopChunk1 %d BottomChunk2 %d TopChunk2 %d \n", row_start, row_count, m_Scale, m_HistoAllocator->m_BottomChunk1, m_HistoAllocator->m_TopChunk1, m_HistoAllocator->m_BottomChunk2, m_HistoAllocator->m_TopChunk2);
		calcHisto<<<row_count, m_NoThreads>>>(row_start, row_count, m_dSourceImage, m_dHalfDiscInfluencePoints, m_TotalHalfInfluencePoints, \
			m_HistoAllocator->m_dHistograms, m_HistoAllocator->m_BottomChunk1, m_HistoAllocator->m_BottomChunk2, m_HistoAllocator->m_TopChunk1, m_HistoAllocator->m_TopChunk2, \
			m_Width, m_Height, m_Scale, m_ArcNo);
		hipDeviceSynchronize();
		m_LastCudaError = hipGetLastError();
		if (m_LastCudaError != hipSuccess) {
			printf("Error execution 1\n");
			return false;
		}
		calculateGradients << <row_count, m_NoThreads >> > (row_start, row_count, m_dGradientImages, \
			m_HistoAllocator->m_dHistograms, m_HistoAllocator->m_BottomChunk1, m_HistoAllocator->m_BottomChunk2, m_HistoAllocator->m_TopChunk1, m_HistoAllocator->m_TopChunk2, \
			m_Width, m_Height, m_Scale, m_ArcNo);
		hipDeviceSynchronize();
		m_LastCudaError = hipGetLastError();
		if (m_LastCudaError != hipSuccess) {
			printf("Error execution 2\n");
			return false;
		}
		for (int k = row_start; k < row_count + row_start; ++k) {
			if (!updateHistoBuffer(m_Step, k))
				return false;
		}	
	}

	m_LastCudaError = hipMemcpy(m_hGradientImages, m_dGradientImages, m_ArcNo * m_Width * m_Height * sizeof(double), hipMemcpyDeviceToHost);
	return m_LastCudaError == hipSuccess;	
}


void CudaPbDetector::readDeviceProperties() {
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	m_SharedMemoryPerBlock = deviceProp.sharedMemPerBlock;
}