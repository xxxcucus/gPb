#include "hip/hip_runtime.h"
#include "cudapbdetector.h"
#include "cvector.h"
#include <cstdlib>


__global__ void calculateGradients(int row_start, int row_count, double* dGradientImages, unsigned int** dHistograms, int image_width, int image_height, int scale, int arcno) {
	int row = row_start + blockIdx.x;
	int index = threadIdx.x;
	int stride = blockDim.x;
	
	if (row < 2 * scale || row >= image_height + 2 * scale || row >= row_start + row_count)
		return;

	for (int j = scale + index; j < image_width + scale; j += stride) {
		unsigned int* vHist = dHistograms[row - scale] +  j * 2 * arcno * 256;

		for (int i = 0; i < arcno; ++i) {
			unsigned int* histo1 = vHist + i * 256;
			unsigned int* histo2 = vHist + (i + arcno) * 256;
			//printf("Chi square for:\n");
			double val = 0.0;
			for (int k = 0; k < 256; ++k) {
				if (histo1[k] != 0 || histo2[k] != 0) {
					double diff = double(int(histo1[k] - histo2[k]));
					double sum = double(histo1[k] + histo2[k]);
					val = val + diff * diff / sum;
					//printf("[%d - %d = %f]", histo1[k], histo2[k], val);
				}

			}
			double grad = val;
			//printf("Grad[%d, %d, %d]=%f\n", row, j, i, grad);
			*(dGradientImages + i * image_width * image_height + (row - 2 * scale) * image_width + (j - scale)) = grad;
		}
	}
}

__global__ void calcHisto(int row_start, int row_count, unsigned char* dSourceImage, struct CVector* dHalfDiscInfluencePoints, int totalHalfInfluencePoints, unsigned int** dHistograms, int image_width, int image_height, int scale, int arcno)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	int i = row_start + blockIdx.x;

	if (blockIdx.x > row_count)
		return;

	for (int j = index; j < image_width + 2 * scale; j += stride) {
		//qDebug() << "BlaBla1 " << j;
		unsigned char val = dSourceImage[i * (image_width + 2 * scale) + j];
		//printf("Index %d Val %d \n", j, int(val));
		//with the point (i,j) with value val, update all histograms which contain this data point
		addToHistoArray(dHalfDiscInfluencePoints, totalHalfInfluencePoints, dHistograms, image_width, image_height, scale, arcno, val, i, j);
	}
}

__device__ void addToHistoArray(struct CVector* dHalfDiscInfluencePoints, int totalHalfInfluencePoints, unsigned int** dHistograms, int image_width, int image_height, int scale, int arcno, int val, int i, int j)
{
	for (int k = 0; k < totalHalfInfluencePoints; ++k) {
		struct CVector n = dHalfDiscInfluencePoints[k];
		if ((n.m_Data[0] + i) < 0 || (n.m_Data[0] + i) >= image_height + 2 * scale)
			continue;
		if ((n.m_Data[1] + j) < 0 || (n.m_Data[1] + j) >= image_width + 2 * scale)
			continue;

		//qDebug() << "Compute at " << i << " with " << (n[0] + i) << " size " << vMaps[n[0] + i].size();
		/*if (int(vMaps[n[0] + i].size()) != m_SingleChannelImage.cols + 2 * m_Scale) {
			qDebug() << "exiting ..";
			exit(1);
		}*/

		unsigned int* vHist = dHistograms[n.m_Data[0] + i] + (n.m_Data[1] + j) * 2 * arcno * 256;
		for (unsigned int l = 2; l < n.m_Size; ++l) {
			if (n.m_Data[l] > 2 * arcno)
				continue;
			//qDebug() << "Insert into histo " << n[k] << " val " << val << " vHist size " << vHist.size();
			//TODO: use atomic operation
			atomicInc(vHist + n.m_Data[l] * 256 + val, 4 * scale * scale);
		}
	}
}

CudaPbDetector::CudaPbDetector(unsigned char* image_data, int image_width, int image_height, int scale) :
	m_Width(image_width), m_Height(image_height), m_Scale(scale)
{
	if (!createGradientImages()) {
		printf("Error in constructor createGradientImages\n");
		return;
	}

	if (!copyImageToGPU(image_data)) {
		printf("Error in constructor copyImageToGPU\n");
		return;
	}

	if (!create2DHistoArray()) {
		printf("Error in constructor create2DHistoArray\n");
		return;
	}

	if (!initializeInfluencePoints()) {
		printf("Error in constructor initializeInfluencePoints\n");
		return;
	}

	m_FullyInitialized = true;
}

bool CudaPbDetector::createGradientImages()
{
	//allocate the device memory for the gradient images
	m_LastCudaError = hipMalloc(&m_dGradientImages, m_ArcNo * m_Width * m_Height * sizeof(double));

	if (m_LastCudaError != hipSuccess) {
		return false;
	}

	//set all pixels in the gradient images to 0
	m_LastCudaError = hipMemset(m_dGradientImages, 0, m_ArcNo * m_Width * m_Height * sizeof(double));

	if (m_LastCudaError != hipSuccess)
		return false;	

	m_hGradientImages = (double*)malloc(m_ArcNo * m_Width * m_Height * sizeof(double));

	return true;
}

bool CudaPbDetector::initializeHistoRange(int start, int stop)
{
	for (int i = start; i < stop; ++i) {
		m_LastCudaError = hipMalloc((void**)&m_hHistograms[i],   256 * 2 * m_ArcNo * (m_Width + 2 * m_Scale) * sizeof(unsigned int));
		//printf("Alloc %d\n", i);
		if (m_LastCudaError != hipSuccess) {
			printf("hipMalloc error 1: %d\n", i);
			return false;
		}
	}

	hipMemcpy(m_dHistograms + start, m_hHistograms + start, (stop - start) * sizeof(unsigned int*), hipMemcpyHostToDevice);
	if (m_LastCudaError != hipSuccess) {
		printf("hipMemcpy error 1\n");
		return false;
	}
	
	return true;
}

bool CudaPbDetector::deleteFromHistoMaps(int step, int index) {

	if (index + m_Scale + step + 1 < m_Height + 2 * m_Scale) {
		if (!initializeHistoRange(index + m_Scale + step + 1, index + m_Scale + step + 2))
			return false;
	}

	if (index >= m_Scale + 1) {
		m_LastCudaError = hipFree(m_hHistograms[index - m_Scale - 1]);
		if (m_LastCudaError != hipSuccess) {
			printf("hipFree error 1: %d\n", index);
			return false;
		}
		//printf("Delete %d\n", index - m_Scale - 1);
		m_hHistograms[index - m_Scale - 1] = nullptr;
	}

	return true;
}

bool CudaPbDetector::create2DHistoArray()
{
	//preparing histograms
	m_LastCudaError = hipMalloc((void**)&m_dHistograms, (m_Height + 2 * m_Scale) * sizeof(unsigned int*));

	if (m_LastCudaError != hipSuccess)
		return false;

	m_hHistograms = (unsigned int**)malloc((m_Height + 2 * m_Scale) * sizeof(unsigned int*));

	return true;
}

//TODO: to check this
CudaPbDetector::~CudaPbDetector()
{
	hipFree(m_dSourceImage);
	hipFree(m_dGradientImages);
	
	for (int i = 0; i < m_Height + 2 * m_Scale; ++i) {
		if (m_hHistograms[i])
			hipFree(m_hHistograms[i]);
	}

	free(m_hHistograms);
	hipFree(m_dHistograms);

	for (int i = 0; i < m_TotalHalfInfluencePoints; ++i) {
		hipFree(m_hHalfDiscInfluencePoints[i].m_Data); 
	}

	free(m_hHalfDiscInfluencePoints);
	hipFree(m_dHalfDiscInfluencePoints);
}

bool CudaPbDetector::copyImageToGPU(unsigned char* image_data)
{
	//copy image to the device memory and pad with zeros
	//TODO: for start pad with zeros, but later as in the CPU method
	//another solution is to pad the image before it is given to this class
	m_LastCudaError = hipMalloc((void**)&m_dSourceImage, (m_Width + 2 * m_Scale) * (m_Height + 2 * m_Scale));

	if (m_LastCudaError != hipSuccess) {
		printf("Error copyImageToGPU hipMalloc %s, %d %d %d\n", hipGetErrorString(m_LastCudaError), m_Width, m_Height, m_Scale);
		//return false;
	}

	//set all pixels in the image to zero 
	m_LastCudaError = hipMemset(m_dSourceImage, 0, (m_Width + 2 * m_Scale) * (m_Height + 2 * m_Scale));

	if (m_LastCudaError != hipSuccess) {
		printf("Error copyImageToGPU hipMemset\n");
		return false;
	}

	//copy from the host image with padding
	int count = 0;
	while (count < m_Height && m_LastCudaError == hipSuccess)
	{
		m_LastCudaError = hipMemcpy(m_dSourceImage + m_Scale * (m_Width + 2 * m_Scale) + count * (m_Width + 2 * m_Scale) + m_Scale, image_data + count * m_Width, m_Width, hipMemcpyHostToDevice);
		if (m_LastCudaError != hipSuccess) {
			printf("Error copyImageToGPU hipMemcpy %d\n", count);
			return false;
		}
		count++;
	}
	
	return true;
}

/**
 * Copies m_Masks->getHalfDiscInfluencePoints()
 * to the GPU
 */
bool CudaPbDetector::initializeInfluencePoints() {
	m_Masks = new DiscInverseMasks(m_Scale);
	std::vector<std::vector<int>> neighb = m_Masks->getHalfDiscInfluencePoints();

	m_TotalHalfInfluencePoints = int(neighb.size());

	m_hHalfDiscInfluencePoints = (CVector*)malloc(m_TotalHalfInfluencePoints * sizeof(CVector));
	for (int i = 0; i < m_TotalHalfInfluencePoints; ++i) {
		m_hHalfDiscInfluencePoints[i].m_Size = int(neighb[i].size());
		m_LastCudaError = hipMalloc(&m_hHalfDiscInfluencePoints[i].m_Data, neighb[i].size() * sizeof(int));
		if (m_LastCudaError != hipSuccess)
			return false;
		int* values = (int*)malloc(neighb[i].size() * sizeof(int));
		
		for (int j = 0; j < neighb[i].size(); ++j)
			values[j] = neighb[i][j];
		m_LastCudaError = hipMemcpy(m_hHalfDiscInfluencePoints[i].m_Data, values, neighb[i].size() * sizeof(int), hipMemcpyHostToDevice);
		if (m_LastCudaError != hipSuccess)
			return false;
		free(values);
	}

//TODO: release memory in case of failure

	//preparing histograms
	m_LastCudaError = hipMalloc(&m_dHalfDiscInfluencePoints, m_TotalHalfInfluencePoints * sizeof(CVector));
	if (m_LastCudaError != hipSuccess)
		return false;

	hipMemcpy(m_dHalfDiscInfluencePoints, m_hHalfDiscInfluencePoints, m_TotalHalfInfluencePoints * sizeof(CVector), hipMemcpyHostToDevice);
	if (m_LastCudaError != hipSuccess)
		return false;

	return true;	
}

bool CudaPbDetector::execute() {
	int noThreads = 256;
	int step = 7;

	hipStream_t stream1;
	hipStream_t stream2;

	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);

	if (!initializeHistoRange(0, m_Scale + 1))
		return false;
	int noSteps = (m_Height + 2 * m_Scale + step - 1) / step;

	/*for (int i = 0; i < noSteps; ++i) {
		int row_start = step * i;
		int row_count = std::min(step, m_Height + 2 * m_Scale - row_start);
		//printf("Row_start: %d Row_count %d \n", row_start, row_count);
		calcHisto<<<row_count, noThreads>>>(row_start, row_count, m_dSourceImage, m_dHalfDiscInfluencePoints, m_TotalHalfInfluencePoints, m_dHistograms, m_Width, m_Height, m_Scale, m_ArcNo);
		hipDeviceSynchronize();
		calculateGradients << <row_count, noThreads >> > (row_start, row_count, m_dGradientImages, m_dHistograms, m_Width, m_Height, m_Scale, m_ArcNo);
		hipDeviceSynchronize();
		for (int k = row_start; k < row_count + row_start; ++k) {
			if (!deleteFromHistoMaps(step, k))
				return false;
		}	
	}*/
	
	int top_allocated = m_Scale + 1;
	int bottom_allocated = 0;

	//new loop
	while (top_allocated < m_Height + 2 * m_Scale) {
		int row_start = top_allocated;
		int row_count = std::min(step, m_Height + 2 * m_Scale - top_allocated);
		//printf("Row_start: %d Row_count %d \n", row_start, row_count);
		
		int new_top_allocated = top_allocated;
		bool allocate_failed = false;

		for (int k = row_start; k < row_count + row_start; ++k) {
			if (k + m_Scale + 1 < m_Height + 2 * m_Scale) {
				if (initializeHistoRange(k + m_Scale + 1, k + m_Scale + 2)) {
					new_top_allocated = k + m_Scale + 1;
				}
				else {
					allocate_failed = true;
					row_count = new_top_allocated - top_allocated;
					break;
				}
			}
		}

		calcHisto << <row_count, noThreads, 0, stream1 >> > (row_start, row_count, m_dSourceImage, m_dHalfDiscInfluencePoints, m_TotalHalfInfluencePoints, m_dHistograms, m_Width, m_Height, m_Scale, m_ArcNo);
		//synchronize in stream
		hipDeviceSynchronize();


		else {
			calcHisto << <row_count, noThreads, 0, stream1 >> > (row_start, row_count, m_dSourceImage, m_dHalfDiscInfluencePoints, m_TotalHalfInfluencePoints, m_dHistograms, m_Width, m_Height, m_Scale, m_ArcNo);
		}
		calculateGradients << <row_count, noThreads >> > (row_start, row_count, m_dGradientImages, m_dHistograms, m_Width, m_Height, m_Scale, m_ArcNo);
		hipDeviceSynchronize();
		for (int k = row_start; k < row_count + row_start; ++k) {
			if (!deleteFromHistoMaps(step, k))
				return false;
		}
	}


	m_LastCudaError = hipMemcpy(m_hGradientImages, m_dGradientImages, m_ArcNo * m_Width * m_Height * sizeof(double), hipMemcpyDeviceToHost);
	return m_LastCudaError == hipSuccess;	
}