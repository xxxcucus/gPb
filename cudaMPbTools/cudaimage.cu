#include "cudaimage.h"

CudaImage::CudaImage(unsigned char* image_data, int image_width, int image_height, int scale) : 
	m_Width(image_width), m_Height(image_height), m_Scale(scale)
{
	//copy image to the device memory and pad with zeros
	//TODO: for start pad with zeros, but later as in the CPU method
	//another solution is to pad the image before it is given to this class
	m_LastCudaError = hipMalloc(&m_dSourceImage, (image_width + 2 * m_Scale) * (image_height + 2 * m_Scale));

	if (m_LastCudaError != hipSuccess)
		return;

	//set all pixels in the image to zero 
	m_LastCudaError = hipMemset(m_dSourceImage, 0, (image_width + 2 * m_Scale) * (image_height + 2 * m_Scale));

	if (m_LastCudaError != hipSuccess)
		return;

	//copy from the host image with padding
	int count = 0;
	while (count < image_height && m_LastCudaError == hipSuccess)
	{
		m_LastCudaError = hipMemcpy(image_data + count * image_width, m_dSourceImage + m_Scale * (image_width + 2 * m_Scale) + count * (image_width + 2 * m_Scale) + m_Scale, image_width, hipMemcpyHostToDevice);
		count++;
	}

	if (m_LastCudaError != hipSuccess)
		return;	

	//allocate the device memory for the gradient images
	m_LastCudaError = hipMalloc(&m_dGradientImages, m_ArcNo * image_width * image_height * sizeof(double));

	if (m_LastCudaError != hipSuccess) {
		m_dGradientImages = nullptr;
		return;
	}

	//set all pixels in the gradient images to 0
	m_LastCudaError = hipMemset(m_dGradientImages, 0, m_ArcNo * image_width * image_height * sizeof(double));

	if (m_LastCudaError != hipSuccess)
		return;

	//preparing histograms
	m_LastCudaError = hipMalloc(&m_dHistograms, (m_Height + 2 * m_Scale) * sizeof(unsigned char*));
	for (int i = 0; i < m_Height + 2 * m_Scale; ++i)
		m_dHistograms[i] = nullptr;

	m_FullyInitialized = true;
}

bool CudaImage::initializeHistoRange(int start, int stop)
{ 
    for (int i = start; i < stop; ++i) {
		m_LastCudaError = hipMalloc(&m_dHistograms[i], 256 * 2 * m_ArcNo * (m_Width + 2 * m_Scale) * sizeof(unsigned char));
		
		if (m_LastCudaError != hipSuccess)
			return false;

		//set all pixels in the gradient images to 0
		m_LastCudaError = hipMemset(m_dHistograms[i], 0, 256 * 2 * m_ArcNo * (m_Width + 2 * m_Scale) * sizeof(unsigned char));

		if (m_LastCudaError != hipSuccess)
			return false;
		}

	return true;
}

void CudaImage::addToHistoMaps(int val, int i, int j)
{
	
}

CudaImage::~CudaImage()
{
	hipFree(m_dSourceImage);
	hipFree(m_dGradientImages);
	
	for (int i = 0; i < m_Height + 2 * m_Scale; ++i)
		if (m_dHistograms[i] != nullptr)
			hipFree(m_dHistograms[i]);

	hipFree(m_dHistograms);
}