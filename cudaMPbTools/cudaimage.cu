#include "cudaimage.h"

CudaImage::CudaImage(unsigned char* image_data, int image_width, int image_height) : 
	m_Width(image_width), m_Height(image_height)
{
	m_LastCudaError = hipMalloc(&m_dSourceImage, image_width * image_height);

	if (m_LastCudaError == hipSuccess) {
		m_LastCudaError = hipMemcpy(image_data, m_dSourceImage, image_width * image_height, hipMemcpyDeviceToHost);
	}
	else {
		m_dSourceImage = nullptr;
		return;
	}

	m_LastCudaError = hipMalloc(&m_GradientImages, m_ArcNo * image_width * image_height * sizeof(double));

	if (m_LastCudaError != hipSuccess)
		return;

	m_LastCudaError = hipMemcpy(m_GradientImages, m_ArcNo * image_width * image_height * sizeof(double));

	if (m_LastCudaError != hipSuccess)
		return;

	m_FullyInitialized = true;
}


CudaImage::~CudaImage()
{
	hipFree(m_dSourceImage);
}